#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <regex>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <random>
#include <cmath>
#include <vector>

// Copy from dense to already allocated bml_tc matrix type 

// if the matrix is not allocated in TC, the allocate and copy 

extern "C"
{
    float *bml_import_mptc (float *);
}

float *
bml_import_mptc (float *A)
{
  int N = 100;
  // Set GPU
  int device = 0;
  hipSetDevice (device);

  // Cublas Handle
  hipblasHandle_t handle;
  hipblasCreate (&handle);

  // Set math mode
  hipblasStatus_t cublasStat =
    hipblasSetMathMode (handle, HIPBLAS_TENSOR_OP_MATH);

  float * A_bml;
  A_bml = (float*)malloc(N * N * sizeof(float));
  printf("bergaberg");
  hipMemcpy (A_bml, A, N * N * sizeof (float), hipMemcpyHostToDevice);

  return A_bml;
}
