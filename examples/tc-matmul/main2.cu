#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <regex>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <random>
#include <cmath>
#include <vector>
#include "tcore_hp_emulator.cuh"
#include "linalg_tools.cuh"

float * bml_multiply_x2_mptc (float *, const int);


__global__ void FtoD(float *X, double *Y, int N) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < N * N) {
    Y[i] = double(X[i]);
    i += blockDim.x * gridDim.x; // add total number of threads to i
  }
}

__global__ void dev_Hamiltonian(float* X, int N){
int j = threadIdx.x + blockIdx.x * blockDim.x; //row number
int i = blockIdx.y; //column number
if (i < N && j < N){

    if (i <= j){
        //printf("(%d,%d) \n",i,j);
        X[i*N+j] = exp(-0.5f*abs((float(i-j))))*sin(float(i+1));
        X[i+N*j] = X[i*N+j];
    }
}
}
__global__ void dev_buildIdenity(float* X, int N){
int i = threadIdx.x + blockIdx.x * blockDim.x; //row number
int j = threadIdx.y + blockIdx.y * blockDim.y; //column number

if (i < N && j < N){
    if (i == j){
        //printf("(%d,%d) \n",i,j);
        X[i*N+j] = 1.0f;
    }else{
        X[i*N+j] = 0.0f;
    }
}
}

void produce_hamiltonian (const unsigned N, float *X) {
    for(int i=0; i<N; ++i) {
        for(int j=i; j<N; ++j) {
            X[i+j*N] = exp(-0.5f*abs((float)(i-j)))*sin((float)(i+1));
            X[j+i*N] = X[i+j*N];
        }
    }
};

double Frobenius (const unsigned N, double *X) {
    double sum=0.0;
    for(int i=0; i<N; ++i) {
        for (int j=0; j<N; j++){
            sum = sum + X[i*N+j]*X[i*N+j];
        }
    }
    return sqrt(sum);
};

void print_Smat (const unsigned m, const unsigned N, float* x) {
     for (int i=0; i<m;i++){
          for (int j=0; j<m; j++){
              std::cout << std::setprecision(15) << x[i*N+j] << " ";
          }
       std::cout << std::endl;
   };
};

int main(int argc, char *argv[])
{

    // Matrix size
    size_t N = atoi(argv[1]);
    size_t Nocc = atoi(argv[2]);

    int Stopp = 0;
    int iter = 0;

    std::vector<float> Idemp_Error;
     
    // Set GPU
    int device = 0;
    hipSetDevice(device);

    // Cublas Handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Set math mode
    hipblasStatus_t cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    
    // Declare Memory,
    double *T, *D, *d_D, *d_TrD, *TrD, *d_T, *d_T2, *d_T4,
           *d_H, *d_energy, *energy, *comm_err, *idem_err, 
           *occ_err, *d_occ_err, *d_idem_err, *d_comm_err, 
           *d_Idd;
    float  *I, *d_S, *d_S2, *d_TrS, *d_TrS2, *d_Sig, *d_Id, 
           *sbuf1, *sbuf2, *TrS, *TrS2, *Sig, *S, *max_eigen;
    half   *hbuf1, *hbuf2;
    int    *v_sgn;
    
    // Allocate host memory
    S = (float*) malloc(N * N * sizeof(float));
    D = (double*) malloc(N * N * sizeof(double));
    I = (float*) malloc(N * N * sizeof(float));
    T = (double*) malloc( N * N * sizeof(double) );
    v_sgn = (int*) malloc( N * sizeof(int) );
    TrS = (float*) malloc(sizeof(float));
    TrS2 = (float*) malloc(sizeof(float));
    Sig = (float*) malloc(sizeof(float));
    max_eigen = (float*) malloc(sizeof(float) );
    TrD = (double*) malloc(sizeof(double) );
    energy = (double*) malloc(sizeof(double));
    comm_err = (double*) malloc(sizeof(double));
    occ_err = (double*) malloc(sizeof(double));
    idem_err = (double*) malloc(sizeof(double));
    

    // Allocate device memory
    hipMalloc(&d_H,N*N*sizeof(double));
    hipMalloc(&d_T,N*N*sizeof(double));
    hipMalloc(&d_T2,N*N*sizeof(double));
    hipMalloc(&d_T4,N*N*sizeof(double));
    hipMalloc(&d_D,N*N*sizeof(double));
    hipMalloc(&d_S,N*N*sizeof(float));
    hipMalloc(&d_S2,N*N*sizeof(float));
    hipMalloc(&d_Id,N*N*sizeof(float));
    hipMalloc(&d_Idd,N*N*sizeof(double));
    hipMalloc(&d_Sig,sizeof(float));
    hipMalloc(&d_TrS,sizeof(float));
    hipMalloc(&d_TrS2,sizeof(float));
    hipMalloc(&d_TrD,sizeof(double));
    hipMalloc(&d_occ_err,sizeof(double));
    hipMalloc(&d_idem_err,sizeof(double));
    hipMalloc(&d_energy,sizeof(double));
    hipMalloc(&d_comm_err,sizeof(double)); 

    // Allocate Buffers
    hipMallocManaged(&sbuf1,  N * N * sizeof(float));
    hipMallocManaged(&sbuf2,  N * N * sizeof(float));
    hipMallocManaged(&hbuf1,  N * N * sizeof(half));
    hipMallocManaged(&hbuf2,  N * N * sizeof(half));
    
    // Define grid size
    int numThreads = 128;
    int numBlocks = N * N / 80 / 128 + 1; 

    // Initialize Hamiltonian and identity
    produce_hamiltonian(N, S);
    hipMemcpy(d_S, S, N*N*sizeof(float), hipMemcpyHostToDevice);
    //dev_Hamiltonian<<<dim3 (N / 32 + 1, N, 1), dim3 (32, 1, 1)>>>(d_S,N);
    dev_buildIdenity<<<dim3 (N, N, 1), dim3 (1, 1, 1)>>>(d_Id, N);

    // Initialize double precision matrices
    FtoD<<<numBlocks,numThreads>>>(d_S,d_H,N);
    FtoD<<<numBlocks,numThreads>>>(d_Id,d_Idd,N);
    
    // Estimate spectral bounds using power method
    //linalgtools::max_eigen(d_S, max_eigen, 1e-15, N, handle);
    float h1=-1.867;//h1 = -abs(max_eigen[0])-1;//-1.867;
    float hN=1.867; //hN = abs(max_eigen[0])+1;//1.867;
    //std::cout << "h1 = " << h1 << std::endl;
    

    // Get device id
    hipGetDevice(&device); 
    
    //compute initial layer of the DNN, W*S+B
    float a = -1/(hN-h1); 
    float b = hN/(hN-h1); 
    cublasStat = hipblasSgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &b,
                             d_Id, N,
                             d_Id, N,  
                             &a,
                             d_S, N);   //this function computes S = b*Id*Id + a*S = W*S + B
     
    // Compute initial trace
    linalgtools::GPUSTrace(N,d_S,d_TrS);
    hipMemcpy(TrS, d_TrS, sizeof(float), hipMemcpyDeviceToHost);  

    // SP2 DNN Loop
    std::cout << "Beginning SP2 DNN..." << std::endl;

    while (Stopp == 0) {
        
        //S^2 - half prec
       d_S2 = bml_multiply_x2_mptc (d_S, N);
	
        // Trace of S^2
        linalgtools::GPUSTrace(N,d_S2,d_TrS2); //only works for N even
        hipMemcpy(TrS2, d_TrS2, sizeof(float), hipMemcpyDeviceToHost); 
    
        // Idempotency error    
        Idemp_Error.push_back(TrS[0]-TrS2[0]);
        std::cout << "Idempotency error = " << Idemp_Error[iter] << std::endl;	
        
        // Convergence control
	if (TrS[0]-TrS2[0]<=0){
            break;
        };
        if (iter>2 && v_sgn[iter-1]!=v_sgn[iter-2]  && Idemp_Error[iter]>= 4.5*Idemp_Error[iter-2]*Idemp_Error[iter-2]){
            break;
        };
        
        // Compute Sigma
        linalgtools::computeSigma(Nocc,d_TrS,d_TrS2,d_Sig);

        // Compute S_{n+1}
        linalgtools::computeSnp1(N*N,d_Sig,d_S2,d_S,d_S);
       
        // Copy traces and sigma to host (seems expensive)
        hipMemcpy(TrS2, d_TrS2, sizeof(float), hipMemcpyDeviceToHost); 
        hipMemcpy(Sig, d_Sig, sizeof(float), hipMemcpyDeviceToHost); 

        // Compute TrS
        TrS[0] = Sig[0]*TrS2[0] + (1-Sig[0])*TrS[0];
        hipMemcpy(d_TrS, TrS, sizeof(float), hipMemcpyHostToDevice); 
        
        // Update sign vector
        v_sgn[iter]=int(Sig[0]);
        
        iter += 1;
    }
    //////////////////////////////////////////////////////
    ////////////// Refinement starts here ////////////////
    //////////////////////////////////////////////////////
    hipDeviceSynchronize();
    FtoD<<<numBlocks, numThreads>>>(d_S, d_T, N);
    //////////////////////////////////////////////////////

    // Compute T^2 in double prec since last update was only to S, not S^2
    double alpha_dbl=1.0, beta_dbl=0.0;
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_T, N,
                             d_T, N,
                             &beta_dbl,
                             d_T2, N); // this function computes T2 = alpha_dbl*T*T + beta_dbl*T2 = T^2 in double precision
    hipDeviceSynchronize();
    hipMemcpy(d_T4, d_T2, N * N * sizeof(double), hipMemcpyDeviceToDevice); 
    
    //////////////////////////////////////////////////////
    ////////////// compute matrix D via GPU //////////////
    ////////////////////////////////////////////////////// 
    alpha_dbl=-1.0,beta_dbl=2.0;
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_T2, N,
                             d_T2, N,
                             &beta_dbl,
                             d_T4, N);  // this function computes D = 2.0*T2 - 1.0*T2*T2 in double precision
    
 
    // Move D to device and host
    hipMemcpy(d_D, d_T4, N * N * sizeof(double), hipMemcpyDeviceToDevice); 
    hipblasGetMatrix(N, N, sizeof(double),
                d_T4, N, D, N);

    //////////////////////////////////////////////////////
    ///////// Compute occupation error via GPU ///////////
    //////////////////////////////////////////////////////
    linalgtools::GPUDTrace(N,d_D,d_TrD); //compute trace on GPU
    hipMemcpy(TrD, d_TrD, sizeof(double), hipMemcpyDeviceToHost);
    occ_err[0] = abs(TrD[0]-Nocc);
    //////////////////////////////////////////////////////'

    //////////////////////////////////////////////////////
    ///////////// Compute energy via GPU /////////////////
    //////////////////////////////////////////////////////
    alpha_dbl = 1.0;
    beta_dbl = 0.0;
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_D, N,
                             d_H, N,
                             &beta_dbl,
                             d_T, N); // set T = D*H
    linalgtools::GPUDTrace(N,d_T,d_energy);
    hipMemcpy(energy, d_energy, sizeof(double), hipMemcpyDeviceToHost);    
    /////////////////////////////////////////////////////// 

    ///////////////////////////////////////////////////////
    ////////// Compute commutation error on GPU ///////////
    ///////////////////////////////////////////////////////
    alpha_dbl=1.0; beta_dbl=-1.0; 
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_H, N,
                             d_D, N,
                             &beta_dbl,
                             d_T, N); // set T = H*D - T = HD-DH   
    //linalgtools::GPUDTrace(N,d_T,d_comm_err);
    //hipMemcpy(comm_err, d_comm_err, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(T, d_T, N * N * sizeof(double), hipMemcpyDeviceToHost);
    comm_err[0] = Frobenius(N,T);     // Commutation error, most sensitive
    //////////////////////////////////////////////////////

    //////////////////////////////////////////////////////
    ///////////// Compute idem err via GPU ///////////////
    //////////////////////////////////////////////////////
    alpha_dbl = 1.0;
    beta_dbl = -1.0;
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_T4, N,
                             d_T4, N,
                             &beta_dbl,
                             d_D, N); // D = D*D-D
    linalgtools::GPUDTrace(N,d_D,d_idem_err);
    hipMemcpy(idem_err, d_idem_err, sizeof(double), hipMemcpyDeviceToHost);
    //hipMemcpy(D, d_D, N*N*sizeof(double), hipMemcpyDeviceToHost);
    //idem_err[0] = Frobenius(N,D);
    /////////////////////////////////////////////////////// 

    // print errors
    std::cout << "Refinement idempotency error: " << std::setprecision(15) << idem_err[0] << std::endl;
    std::cout << "Refinement occupation error: " << std::setprecision(15) << occ_err[0] << std::endl;
    std::cout << "Refinement commutation error: " << std::setprecision(15) << comm_err[0] << std::endl;
    std::cout << "Post-refinement energy: " << energy[0] << std::endl; 
    
    //Deallocate device memory
    hipFree(d_H);
    hipFree(d_S);
    hipFree(d_S2);
    hipFree(d_T);
    hipFree(d_T2);
    hipFree(d_T4);
    hipFree(d_D);
    hipFree(d_Id);
    hipFree(d_Idd);
    hipFree(d_TrD);
    hipFree(d_Sig);
    hipFree(d_TrS);
    hipFree(d_TrS2);
    hipFree(d_idem_err);
    hipFree(d_energy);
    hipFree(d_comm_err);
    hipFree(sbuf1);
    hipFree(sbuf2);
    hipFree(hbuf1);
    hipFree(hbuf2);



    //Deallocate host memory
    free(D);
    free(T);
    free(v_sgn);
    free(TrD);
    free(TrS);
    free(TrS2);
    free(Sig);
    free(energy);
    free(comm_err);
    free(occ_err);
    free(idem_err);
 
    // Destroy handle
    hipblasDestroy(handle);

    return 0;
}



