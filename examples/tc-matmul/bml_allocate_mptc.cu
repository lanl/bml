// Create a zero matrix in GPU 

#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <regex>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <random>
#include <cmath>
#include <vector>
#include "tcore_hp_emulator.cuh"


bml_zero_matrix_dense_mptc (N)
     int main (int argc, char *argv[])
{


  // Set GPU
  int device = 0;
  hipSetDevice (device);

  // Declare Memory,
  float /* *sbuf1, *sbuf2,*/ *S, *S2, *d_S, *d_S2;
  //half *hbuf1, *hbuf2;

  // Allocate host memory
  S = (float *) malloc (N * N * sizeof (float));
  S2 = (float *) malloc (N * N * sizeof (float));


  // Allocate device memory
  hipMalloc (&d_S, N * N * sizeof (float));
  hipMalloc (&d_S2, N * N * sizeof (float));

  // Allocate Buffers
  //hipMallocManaged (&sbuf1, N * N * sizeof (float));
  //hipMallocManaged (&sbuf2, N * N * sizeof (float));
  //hipMallocManaged (&hbuf1, N * N * sizeof (half));
  //hipMallocManaged (&hbuf2, N * N * sizeof (half));


  return 0;
}
