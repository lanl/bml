// Create a zero matrix in GPU 

#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <regex>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <random>
#include <cmath>
#include <vector>
#include "tcore_hp_emulator.cuh"



int bml_deallocate_mptc (hipblasHandle_t handle, float * A)
{


  hipblasDestroy(handle);
  hipFree(A);
  
  return 0;
};
