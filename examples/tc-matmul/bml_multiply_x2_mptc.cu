#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <random>
#include <ctime>


// Device function for splitting a single into two halves
__device__ void
split_single (const float x, half & hi, half & lo)
{
  hi = __float2half (x);
  float y = (x - __half2float (hi));
  lo = __float2half (y * 1024.0);
}

template < typename T > __global__ void
array_split_single (const float *AF, T * AH1, T * AH2, const unsigned N)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N)
    {
      half hi;
      half lo;

      split_single (AF[i], hi, lo);

      //AH1[i] = __half2float(hi);
      //AH2[i] = __half2float(lo);
      AH1[i] = hi;
      AH2[i] = lo;
    }
}

void
tcoreSPGemmSymm (hipblasHandle_t & handle, const unsigned N, const float *A,
		 half * Ah, half * Al, float *B1, float *B2, float *B)
{
  // Setup kernel launch
  unsigned MAX_THREADS = 1024;
  unsigned BLOCKS = ceil (N * N / float (MAX_THREADS));
  unsigned THREADS = MAX_THREADS;

  // Split the floats into the high and low parts
  array_split_single < half ><<< BLOCKS, THREADS >>> (A, Ah, Al, N * N);

  // Set the math mode to allow cuBLAS to use Tensor Cores:
  hipblasStatus_t cublasStat =
    hipblasSetMathMode (handle, HIPBLAS_TENSOR_OP_MATH);

  float alpha (1.0f);
  float beta (0.0f);

  // Compute gemm for high
  cublasStat =
    hipblasGemmEx (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, Ah,
		  HIP_R_16F, N, Ah, HIP_R_16F, N, &beta, B1, HIP_R_32F, N,
		  HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  // Compute gemm for low
  cublasStat =
    hipblasGemmEx (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, Ah,
		  HIP_R_16F, N, Al, HIP_R_16F, N, &beta, B2, HIP_R_32F, N,
		  HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  alpha = 1.0f;
  beta = 1.0f;
  cublasStat = hipblasSgeam (handle,
			    HIPBLAS_OP_N, HIPBLAS_OP_T,
			    N, N, &alpha, B2, N, &beta, B2, N, B, N);

  beta = powf (2, -10);
  cublasStat = hipblasSgeam (handle,
			    HIPBLAS_OP_N, HIPBLAS_OP_N,
			    N, N, &alpha, B1, N, &beta, B, N, B, N);
};



float *
bml_multiply_x2_mptc (float *X, const int N)
{
  // Cublas Handle
  hipblasHandle_t handle;
  hipblasCreate (&handle);

  // Declare Memory,
  float *sbuf1, *sbuf2, *X2;
  half *hbuf1, *hbuf2;

  // Allocate Buffers
  hipMalloc (&X2, N * N * sizeof (float));
  hipMalloc (&sbuf1, N * N * sizeof (float));
  hipMalloc (&sbuf2, N * N * sizeof (float));
  hipMalloc (&hbuf1, N * N * sizeof (half));
  hipMalloc (&hbuf2, N * N * sizeof (half));


  //S^2 - half prec
  tcoreSPGemmSymm (handle, N, X, hbuf1, hbuf2, sbuf1, sbuf2, X2);

  //Deallocate device memory
  hipFree (sbuf1);
  hipFree (sbuf2);
  hipFree (hbuf1);
  hipFree (hbuf2);

  return X2;
}
