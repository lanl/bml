#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <regex>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <random>
#include <ctime>
#include <cmath>
#include <vector>
#include <chrono>
#include "../include/tcore_hp_emulator.cuh"
#include "../include/linalg_tools.cuh"



double Frobenius (const unsigned N, double *X) {
    double sum=0.0;
    for(int i=0; i<N; ++i) {
        for (int j=0; j<N; j++){
            sum = sum + X[i*N+j]*X[i*N+j];
        }
    }
    return sqrt(sum);
};

float trace (const unsigned N,const float *A) {
    float sum (0.0f);
    for (unsigned i = 0; i < N; ++i) {
        sum += A[i+i*N];
    }
    return sum;
};

float Dtrace (const unsigned N, const double *A) {
    double sum=0.0;
    for (unsigned i = 0; i < N; ++i) {
        sum += A[i+i*N];
    }
    return sum;
};

void produce_hamiltonian (const unsigned N, float *X) {
    for(int i=0; i<N; ++i) {
        for(int j=i; j<N; ++j) {
            X[i+j*N] = exp(-0.5f*abs((float)(i-j)))*sin((float)(i+1));
            X[j+i*N] = X[i+j*N];
        }
    }
};

void CPU_float_to_double(float* S, double* T, int N) {
    for(size_t i=0; i<N; ++i) {
        for(size_t j=0; j<N; ++j) {
            T[i+j*N] = double(S[i+j*N]);
        }
    }
};

void CPU_double_to_float(double* S, float *T, int N) {
    for(size_t i=0; i<N; ++i) {
        for(size_t j=0; j<N; ++j) {
            T[i+j*N] = float(S[i+j*N]);
        }
    }
};
void matmul(double xi, double* X, double* Y, double zeta, double* Z, int N) {
    //double* z;
    //z = (double*) malloc(N*N*sizeof(double));
    for(size_t i=0; i<N; ++i) {
        for(size_t j=0; j<N; ++j) {
        double local_sum = 0.0;    
            for (size_t k=0; k<N;k++){
            	local_sum += xi*X[i*N+k]*Y[k*N+j];
            }
        Z[i*N+j] = local_sum+zeta*Z[i*N+j];
        }
    }
    for(size_t i=0; i<N; ++i) {
        for(size_t j=0; j<N; ++j) {
            //Z[i*N+j] = z[i*N+j];
        }
    }
    
};

void build_identity (const unsigned N, float *X) {

    for(size_t i=0; i<N; ++i) {
        for(size_t j=0; j<N; ++j) {
            if (i==j){
                X[i*N+j] = 1;
            }else{
                X[i*N+j] = 0;
            }
        }
    }
};

void print_mat (const unsigned m, const unsigned n, double* x) {
    for (int i=n-m; i<n;i++){
         for (int j=n-m; j<n; j++){
             std::cout << std::setprecision(15) << x[j*n+i] << " ";
         }
         std::cout << std::endl;
     };
};
void print_Smat (const unsigned m, const unsigned n, float* x) {
    for (int i=n-m; i<n;i++){
         for (int j=n-m; j<n; j++){
             std::cout << std::setprecision(15) << x[i*n+j] << " ";
         }
         std::cout << std::endl;
     };
};

void print_mat2csv (const unsigned n, const double *x){
      std::ofstream myfile;
      myfile.open ("mat.csv");
      for (int i=0; i<n;i++){
         for (int j=0; j<n; j++){
             if (j<n-1){
                 myfile << x[i*n+j] << ", ";
             }else{
                 myfile << x[i*n+j];
             };
         };
         myfile << "\n";
     };
     myfile.close();}



int main(int argc, char *argv[])
{

    // Matrix size
    size_t N = atoi(argv[1]);
    size_t Nocc = atoi(argv[2]);

    int Stopp = 0;
    int Kvot = 0;
    int iter = 0;
    int Pur_Start = 0;
    float eps = 1e-16;

    // Prior estimate lower spectral bound
    float h1 = -1.867;
    float hN = 1.867;

    std::vector<float> Idemp_Error;
    
    std::cout << "Mat Size: " << N << std::endl;
    std::cout << "Occupied orbitals: " << Nocc << std::endl;

    // Set GPU
    int device = 0;
    hipSetDevice(device);

    // Cublas Handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    

    hipblasStatus_t cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);


    // Declare Memory,
    double *T, *d_T, *d_T2, *d_T4, *TrT, *TrT2, *D, *D2, *d_D, *D_temp, *d_TrD,*TrD, *TrD2, *d_H, *H, *Idd, *d_Idd, *d_energy, *energy, *comm_err, *idem_err, *occ_err, *d_comm_err;
    float *d_Hs, *S, *d_S, *S2, *Id, *sbuf1, *sbuf2, *sbuf3, *sbuf4, *TrS, *TrSOld, *TrS2, *Sig, *d_senergy;
    half *hbuf1, *hbuf2;
    int *v_sgn;

    float b = hN/(hN-h1); 
    float a = -1/(hN-h1);
 
    // Allocate Memory
    hipMallocManaged(&S,     N * N * sizeof(float));
    hipMallocManaged(&S2,     N * N * sizeof(float));
    v_sgn = (int*) malloc(N*sizeof(int));
    T = (double*) malloc(N*N*sizeof(double));
    hipMalloc(&d_T,N*N*sizeof(double));
    hipMalloc(&d_T2,N*N*sizeof(double));
    hipMalloc(&d_T4,N*N*sizeof(double));
    D = (double*) malloc(N*N*sizeof(double));
    hipMalloc(&d_D,N*N*sizeof(double));
    Idd = (double*) malloc(N*N*sizeof(double));
    hipMalloc(&d_Idd,N*N*sizeof(double));
    H = (double*) malloc(N*N*sizeof(double));
    hipMalloc(&d_H,N*N*sizeof(double));
    hipMalloc(&d_Hs,N*N*sizeof(float));
    hipMalloc(&d_S,N*N*sizeof(float));
    TrD = (double*) malloc(sizeof(double));
    hipMalloc(&d_TrD,sizeof(double));
    energy = (double*) malloc(sizeof(double));
    hipMalloc(&d_energy,sizeof(double));
    hipMalloc(&d_senergy,sizeof(float));
    comm_err = (double*) malloc(sizeof(double));
    hipMalloc(&d_comm_err,sizeof(double));
    

    //hipMallocManaged(&D,     N * N * sizeof(double));
    hipMallocManaged(&D_temp,     N * N * sizeof(double)); 
    hipMallocManaged(&D2,     N * N * sizeof(double));    
    hipMallocManaged(&Id,     N * N * sizeof(float));
    hipMallocManaged(&TrS,    sizeof(float));
    hipMallocManaged(&TrS2,    sizeof(float));
    hipMallocManaged(&TrT,    sizeof(double));
    hipMallocManaged(&TrT2,    sizeof(double));
    hipMallocManaged(&TrD2,    sizeof(double));
    hipMallocManaged(&TrSOld,    sizeof(float));
    hipMallocManaged(&Sig,    sizeof(float));
    hipMallocManaged(&occ_err,    sizeof(double));
    hipMallocManaged(&idem_err,    sizeof(double));

    // Allocate Buffers
    hipMallocManaged(&sbuf1,  N * N * sizeof(float));
    hipMallocManaged(&sbuf2,  N * N * sizeof(float));
    hipMallocManaged(&sbuf3,  N * N * sizeof(float));
    hipMallocManaged(&sbuf4,  N * N * sizeof(float));
    hipMallocManaged(&hbuf1,  N * N * sizeof(half));
    hipMallocManaged(&hbuf2,  N * N * sizeof(half));

    // Produce Hamiltonian and Identity matrix 
    std::cout << "Loading Hamiltonian..." << std::endl;
    produce_hamiltonian(N,S);
    hipMemcpy(d_Hs, S, N * N * sizeof(float), hipMemcpyHostToDevice); // Send H to d_H   
    
    CPU_float_to_double(S,H,N); //change hamiltonian to double precision
    hipMemcpy(d_H, H, N * N * sizeof(double), hipMemcpyHostToDevice); // Send H to d_H  
    build_identity(N,Id);
    CPU_float_to_double(Id,D,N);
    
    // Get device id
    hipGetDevice(&device); 

    
    //compute initial layer of the DNN, W*S+B
    
    cublasStat = hipblasSgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &b,
                             Id, N,
                             Id, N,  
                             &a,
                             S, N);   //this function computes S = b*Id*Id + a*S = W*S + B
     
    //std::cout << S[0]<<" "<< S[1]<< " " << S[2]<<std::endl;

    // Compute initial trace
    linalgtools::GPUSTrace2(N,S,TrS);
    
    // begin timing
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    

    // SP2 DNN Loop
    std::cout << "Beginning SP2 DNN..." << std::endl;
    while (Stopp == 0) {
        hipMemPrefetchAsync(TrS,   sizeof(float), device, NULL);
        hipDeviceSynchronize();

        //S^2 - half prec
        tcoretools::tcoreSPGemmSymm(handle
                                   ,N
                                   ,S
                                   ,hbuf1
                                   ,hbuf2
                                   ,sbuf1
                                   ,sbuf2
                                   ,S2);

	// Trace of S^2
        linalgtools::GPUSTrace2(N,S2,TrS2);
        hipMemPrefetchAsync(TrS2,   sizeof(float), device, NULL);
        hipDeviceSynchronize();
        Idemp_Error.push_back(TrS[0]-TrS2[0]);
        std::cout << "Idempotency error = " << Idemp_Error[iter] << std::endl;	
        
        // Convergence Control
	if (TrS[0]-TrS2[0]<=0){
            break;
        };
        if (iter>2 && v_sgn[iter-1]!=v_sgn[iter-2]  && Idemp_Error[iter]>= 4.5*Idemp_Error[iter-2]*Idemp_Error[iter-2]){
            break;
        };

/*      if (abs(TrS2[0]-TrS[0]) < eps) {
            std::cout <<  "Converged!" << std::endl;
            Stopp = 1;
            iter -=1;
        } else {
            Idemp_Error.push_back(abs(TrS2[0]-TrS[0]) + eps);
           // std::cout << iter+1 << ") IdErr=" << Idemp_Error[iter] << std::endl;
            if (iter > 1) {
                Kvot = Idemp_Error[iter-2]/Idemp_Error[iter];
                if (abs(TrS2[0] - Nocc) < 0.1) {
                    if ((Pur_Start == 0) && (Kvot > 4)) {
                        Pur_Start = 1;
                    } else if ((Pur_Start == 1) && (Kvot < 3)) {
                        Stopp = 1;
                    }
                }
            }
        } 
*/
        // Compute Sigma
        linalgtools::computeSigma(Nocc,TrS,TrS2,Sig);
        
        // Compute S_{n+1}
        linalgtools::computeSnp1(N*N,Sig,S2,S,S);
        hipDeviceSynchronize();
        
        // Compute TrS
        TrS[0] = Sig[0]*TrS2[0] + (1-Sig[0])*TrS[0];
        std::cout << TrS[0] << std::endl; 
        // Update sign vector
        v_sgn[iter]=int(Sig[0]);
        
        hipMemPrefetchAsync(TrS, sizeof(float), device, NULL);
        iter += 1;
        
    }
    // Compute timing of loop
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    double duration = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    std::cout << "Time difference = " << duration << "[µs]" << std::endl;
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::milliseconds> (end - begin).count() << "[ms]" << std::endl;
    double time = 2*iter*N*N*N/(duration/1e6);
    std::cout << time << std::endl;
    

    //////////////////////////////////////////////////////
    ////////////// Refinement starts here ////////////////
    //////////////////////////////////////////////////////
    hipDeviceSynchronize();
    CPU_float_to_double(S,T,N);
    CPU_float_to_double(Id,Idd,N); 

    //////////////////////////////////////////////////////
    //// Send double precision object back to the GPU ///
    //////////////////////////////////////////////////////
    hipMemcpy(d_T, T, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Idd, Idd, N * N * sizeof(double), hipMemcpyHostToDevice); 
    //////////////////////////////////////////////////////
    
    //record time to file
    std::ofstream myfile;
    myfile.open ("timings_A100.csv", std::ios::app);
    myfile << N << ", " << time << "\n";
    myfile.close();


    // Compute T^2 in double prec since last update was only to S, not S^2
    double alpha_dbl=1.0, beta_dbl=0.0;
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_T, N,
                             d_T, N,
                             &beta_dbl,
                             d_T2, N); // this function computes T2 = alpha_dbl*T*T + beta_dbl*T2 = T^2 in double precision
    hipDeviceSynchronize();
    hipMemcpy(d_T4, d_T2, N * N * sizeof(double), hipMemcpyDeviceToDevice); 

    //////////////////////////////////////////////////////
    ////////////// compute matrix D via GPU //////////////
    ////////////////////////////////////////////////////// 
    alpha_dbl=-1.0,beta_dbl=2.0;
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_T2, N,
                             d_T2, N,
                             &beta_dbl,
                             d_T4, N);  // this function computes D = 2.0*T2 - 1.0*T2*T2 in double precision
    hipMemcpy(d_D, d_T4, N * N * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(D, d_T4, N * N * sizeof(double), hipMemcpyDeviceToHost);
    

    //////////////////////////////////////////////////////
    ///////// Compute occupation error via GPU ///////////
    //////////////////////////////////////////////////////
    linalgtools::GPUDTrace(N,d_D,d_TrD); //compute trace on GPU
    hipMemcpy(TrD, d_TrD, sizeof(double), hipMemcpyDeviceToHost);
    occ_err[0] = abs(TrD[0]-Nocc);
    //std::cout << "occ error tr(D) = " << std::setprecision(15) << TrD[0] << std::endl; 
    //////////////////////////////////////////////////////'

    //////////////////////////////////////////////////////
    ///////////// Compute energy via GPU /////////////////
    //////////////////////////////////////////////////////
    alpha_dbl = 1.0;
    beta_dbl = 0.0;
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_D, N,
                             d_H, N,
                             &beta_dbl,
                             d_T, N); // set T = D*H
    hipMemcpy(T, d_T, N*N*sizeof(double), hipMemcpyDeviceToHost);    
    linalgtools::GPUDTrace(N,d_T,d_energy);
    hipMemcpy(energy, d_energy, sizeof(double), hipMemcpyDeviceToHost);    
    /////////////////////////////////////////////////////// 


    ///////////////////////////////////////////////////////
    ////////// Compute commutation error on GPU ///////////
    ///////////////////////////////////////////////////////
    comm_err[0]=1.0;
    alpha_dbl=-1.0; beta_dbl=1.0; 
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_H, N,
                             d_D, N,
                             &beta_dbl,
                             d_T, N); // set T = H*D - T = HD-DH   

    hipMemcpy(T, d_T, N * N * sizeof(double), hipMemcpyDeviceToHost);
    comm_err[0] = Frobenius(N,T);     // Commutation error, most sensitive
    ///////////////////////////////////////////////////////
    

    //////////////////////////////////////////////////////
    ///////////// Compute idem err via GPU ///////////////
    //////////////////////////////////////////////////////
    alpha_dbl = 1.0;
    beta_dbl = -1.0;
    cublasStat = hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha_dbl,
                             d_T4, N,
                             d_T4, N,
                             &beta_dbl,
                             d_D, N); // D = D*D-D
    hipMemcpy(D, d_D, N*N*sizeof(double), hipMemcpyDeviceToHost);
    idem_err[0] = Frobenius(N,D);
    /////////////////////////////////////////////////////// 

    // print errors
    std::cout << "Refinement idempotency error: " << std::setprecision(15) << idem_err[0] << std::endl;
    std::cout << "Refinement occupation error: " << std::setprecision(15) << occ_err[0] << std::endl;
    std::cout << "Refinement commutation error: " << std::setprecision(15) << comm_err[0] << std::endl;
    std::cout << "Post-refinement energy: " << energy[0] << std::endl; 
    return 0;
}



